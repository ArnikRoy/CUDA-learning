#include <stdio.h>
#include ""

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Number of CUDA enabled devices: %d\n", deviceCount);
    hipDeviceProp_t dProps;
    hipError_t e;
    for (int i = 0; i < deviceCount; i++) {
        e = hipGetDeviceProperties(&dProps, i);
        if (e == hipSuccess) {
            printf("For device %d:\n", i);
            printf("Max. threads per block: %d\n", dProps.maxThreadsPerBlock);
            printf("#SM: %d\n", dProps.multiProcessorCount);
            printf("Maximum #blocks per SM: %d\n", dProps.maxBlocksPerMultiProcessor);
            printf("Threads per SM: %d\n", dProps.maxThreadsPerMultiProcessor);
            printf("Warp size in threads: %d\n", dProps.warpSize);
            printf("#Registers available per block/SM: %d\n", dProps.regsPerBlock);
            printf("Clock rate: %d\n", dProps.clockRate);
        } else {
            printf("Error getting properties for device %d!\n", i);
        }
        printf("\n");
    }
}